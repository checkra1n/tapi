
#include <hip/hip_runtime.h>
// Checks errors generated by passing a bad value for --cuda-gpu-arch.
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=compute_20 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix BAD %s
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm20 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix BAD %s
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_19 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix BAD %s
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=gfx900 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix BAD %s

// BAD: error: unsupported CUDA gpu architecture

// RUN: %clang -### -v --target=x86_64-linux-gnu --cuda-gpu-arch=sm_21 \
// RUN:   --cuda-path=%S/Inputs/CUDA_90/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefix BAD_CUDA9 %s

// BAD_CUDA9: GPU arch sm_21 is supported by CUDA versions between 7.0 and 8.0

// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_20 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix OK %s
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_52 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix OK %s
// RUN: %clang -### -x hip -target x86_64-linux-gnu --cuda-gpu-arch=gfx908 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix OK %s
// RUN: %clang -### -x hip -target x86_64-linux-gnu --cuda-gpu-arch=gfx90a -c %s 2>&1 \
// RUN: | FileCheck -check-prefix OK %s
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=gfx940 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix OK %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix OK %s

// We don't allow using NVPTX/AMDGCN for host compilation.
// RUN: %clang -### --cuda-host-only -target nvptx-nvidia-cuda -c %s 2>&1 \
// RUN: | FileCheck -check-prefix HOST_NVPTX %s
// RUN: %clang -### --cuda-host-only -target amdgcn-amd-amdhsa -c %s 2>&1 \
// RUN: | FileCheck -check-prefix HOST_AMDGCN %s

// OK-NOT: error: Unsupported CUDA gpu architecture
// HOST_NVPTX: error: unsupported architecture 'nvptx' for host compilation
// HOST_AMDGCN: error: unsupported architecture 'amdgcn' for host compilation
